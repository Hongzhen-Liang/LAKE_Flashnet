#include "hip/hip_runtime.h"
#include<stdio.h>
#include <stdbool.h> 
#include "weights.h"
#include<sys/time.h>
#define LEN_INPUT 31
#define LEN_LAYER_0 256
#define LEN_LAYER_0_HALF 128
#define LEN_LAYER_1 2
#define FEAT_31
#define NUM_PARALLEL 16


__global__ void prediction_mid_layer(long *weight_0_T_ent, long *bias_0_ent, long *input_vec_i, long *mid_res_i) { 
	int j, offset;

	int threadId = threadIdx.x;
    int stride = blockDim.x;
	int input_ind = blockIdx.x*LEN_INPUT;
	int blockId = blockIdx.x;
	for (j = threadId, offset=threadId*LEN_INPUT; j < LEN_LAYER_0; j+=stride, offset+=LEN_INPUT*stride) {
		int update_index = blockId*stride + j;
        mid_res_i[update_index] = 0;
		//loop unroll
		mid_res_i[update_index] =  mid_res_i[update_index] + input_vec_i[input_ind + 0] * weight_0_T_ent[offset+0]
		+ input_vec_i[input_ind + 1] * weight_0_T_ent[offset+1]
		+ input_vec_i[input_ind + 2] * weight_0_T_ent[offset+2]
		+ input_vec_i[input_ind + 3] * weight_0_T_ent[offset+3]
		+ input_vec_i[input_ind + 4] * weight_0_T_ent[offset+4]
		+ input_vec_i[input_ind + 5] * weight_0_T_ent[offset+5]
		+ input_vec_i[input_ind + 6] * weight_0_T_ent[offset+6]
		+ input_vec_i[input_ind + 7] * weight_0_T_ent[offset+7]
		+ input_vec_i[input_ind + 8] * weight_0_T_ent[offset+8]
		+ input_vec_i[input_ind + 9] * weight_0_T_ent[offset+9]
		+ input_vec_i[input_ind + 10] * weight_0_T_ent[offset+10]
		+ input_vec_i[input_ind + 11] * weight_0_T_ent[offset+11]
		+ input_vec_i[input_ind + 12] * weight_0_T_ent[offset+12]
		+ input_vec_i[input_ind + 13] * weight_0_T_ent[offset+13]
		+ input_vec_i[input_ind + 14] * weight_0_T_ent[offset+14]
		+ input_vec_i[input_ind + 15] * weight_0_T_ent[offset+15]
		+ input_vec_i[input_ind + 16] * weight_0_T_ent[offset+16]
		+ input_vec_i[input_ind + 17] * weight_0_T_ent[offset+17]
		+ input_vec_i[input_ind+ 18] * weight_0_T_ent[offset+18]
		+ input_vec_i[input_ind + 19] * weight_0_T_ent[offset+19]
		+ input_vec_i[input_ind + 20] * weight_0_T_ent[offset+20]
		+ input_vec_i[input_ind + 21] * weight_0_T_ent[offset+21]
		+ input_vec_i[input_ind + 22] * weight_0_T_ent[offset+22]
		+ input_vec_i[input_ind + 23] * weight_0_T_ent[offset+23]
		+ input_vec_i[input_ind + 24] * weight_0_T_ent[offset+24]
		+ input_vec_i[input_ind + 25] * weight_0_T_ent[offset+25]
		+ input_vec_i[input_ind + 26] * weight_0_T_ent[offset+26]
		+ input_vec_i[input_ind + 27] * weight_0_T_ent[offset+27]
		+ input_vec_i[input_ind + 28] * weight_0_T_ent[offset+28]
		+ input_vec_i[input_ind + 29] * weight_0_T_ent[offset+29]
		+ input_vec_i[input_ind + 30] * weight_0_T_ent[offset+30];

        // apply bias
        mid_res_i[update_index] += bias_0_ent[threadId];
        // relu
        if (mid_res_i[update_index] < 0) {
            mid_res_i[update_index] = 0;
        }		
    }
}

__global__ void prediction_final_layer(long *weight_1_T_ent, long *bias_1_ent, long *mid_res_i, long *final_res_i) {
    
	int index = blockIdx.x;
	int threadId = threadIdx.x;
	int dim = blockDim.x;
	int k;
	int update_index = index*dim + threadId;
	if (threadId < 32) {
		final_res_i[update_index] = 0;
		for(k = threadId; k<LEN_LAYER_0; k = k + 32) {
			final_res_i[update_index] =  final_res_i[update_index] + mid_res_i[index*LEN_LAYER_0 + k] * weight_1_T_ent[k];
		}
	} else {
		final_res_i[update_index] = 0;
		for(k = threadId - 32; k<LEN_LAYER_0; k = k + 32) {
			final_res_i[update_index] =  final_res_i[update_index] + mid_res_i[index*LEN_LAYER_0 + k] * weight_1_T_ent[k+256];
		}
	}
	__syncthreads();
	if (threadId == 0) {
		update_index = index*dim;
		for(int i = 1; i < 32; i++) {
			final_res_i[update_index] = final_res_i[update_index] + final_res_i[update_index + i];
		}
		final_res_i[update_index] =  final_res_i[update_index] + bias_1_ent[0];
	}
	if(threadId == 32) {
		update_index = index*dim + 32;
		for(int i = 1; i < 32; i++) {
			final_res_i[update_index] = final_res_i[update_index] + final_res_i[update_index + i];
		} 
		final_res_i[update_index] =  final_res_i[update_index] + bias_1_ent[1];
	}
}

static void prediction_model(long *d_input_vec_i, long *d_weight_0_T_ent, 
			long *d_weight_1_T_ent, long *d_bias_0_ent, long *d_bias_1_ent, long *d_mid_res_i, long *d_final_res_i, bool *res) {

	long final_res_i[NUM_PARALLEL*64];

	prediction_mid_layer<<<NUM_PARALLEL,256>>>(d_weight_0_T_ent, d_bias_0_ent, d_input_vec_i, d_mid_res_i);
	prediction_final_layer<<<NUM_PARALLEL, 64>>>(d_weight_1_T_ent, d_bias_1_ent, d_mid_res_i, d_final_res_i);

	hipMemcpy(final_res_i, d_final_res_i, sizeof(long) * 64 * NUM_PARALLEL, hipMemcpyDeviceToHost);
	for(int i = 0; i < NUM_PARALLEL; i++) {
		// printf("\n %ld", final_res_i[i*64]);
		// printf("\n %ld", final_res_i[i*64 + 32]);
		res[i] = final_res_i[i*64]>=(final_res_i[i *64 + 32])? false: true;
	}
	
}

int main() {
	long *weight_0_T_ent, * bias_0_ent, *weight_1_T_ent, * bias_1_ent; 
	long input_vec_i[31] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,0,0,0,0,9,0,0,0,9,0,0,0,9};
	long parallel_input[NUM_PARALLEL][31];
	for(int i = 0 ; i < NUM_PARALLEL; i++) {
		for(int j = 0; j < 31; j++)
			parallel_input[i][j] = input_vec_i[j];
	}

	weight_0_T_ent = &weight_i_0_T[0][0];
	weight_1_T_ent = &weight_i_1[0][0];
	bias_0_ent = bias_i_0;
	bias_1_ent = bias_i_1;

	long *d_weight_0_T_ent, *d_weight_1_T_ent, *d_bias_0_ent, *d_bias_1_ent, *d_input_vec_i, *d_mid_res_i, *d_final_res_i;

	
	hipMalloc((void**)&d_weight_0_T_ent, sizeof(long) * 256*31);
	hipMalloc((void**)&d_weight_1_T_ent, sizeof(long) * 256*2);
	hipMalloc((void**)&d_bias_0_ent, sizeof(long) * 256);
	hipMalloc((void**)&d_bias_1_ent, sizeof(long) *2);

	hipMalloc((void**)&d_mid_res_i, sizeof(long) *LEN_LAYER_0 * NUM_PARALLEL);
	hipMalloc((void**)&d_final_res_i, sizeof(long) *LEN_LAYER_1 * NUM_PARALLEL *32);
	bool res[NUM_PARALLEL];

	
	hipMalloc((void**)&d_input_vec_i, sizeof(long) *LEN_INPUT * NUM_PARALLEL);
	hipMemcpy(d_weight_0_T_ent, weight_0_T_ent, sizeof(long) * 256*31, hipMemcpyHostToDevice);
	hipMemcpy(d_weight_1_T_ent, weight_1_T_ent, sizeof(long) * 256*2, hipMemcpyHostToDevice);
	hipMemcpy(d_bias_0_ent, bias_0_ent, sizeof(long) * 256, hipMemcpyHostToDevice);
	hipMemcpy(d_bias_1_ent, bias_1_ent, sizeof(long) * 2, hipMemcpyHostToDevice);
	struct timeval tval_before, tval_after, tval_result;
	gettimeofday(&tval_before, NULL);
	
	for(int i = 0; i < 1; i++) {
		hipMemcpy(d_input_vec_i, parallel_input, sizeof(long) * LEN_INPUT * NUM_PARALLEL, hipMemcpyHostToDevice);
		 prediction_model(d_input_vec_i, d_weight_0_T_ent, 
			d_weight_1_T_ent, d_bias_0_ent, d_bias_1_ent, d_mid_res_i, d_final_res_i, res);
	}
	
	gettimeofday(&tval_after, NULL);
	timersub(&tval_after, &tval_before, &tval_result);
	printf("Time elapsed: %ld.%06ld\n", (long int)tval_result.tv_sec, (long int)tval_result.tv_usec);

	hipFree(d_input_vec_i);
	hipFree(d_weight_0_T_ent);
	hipFree(d_weight_1_T_ent);
	hipFree(d_bias_0_ent);
	hipFree(d_bias_1_ent);
	hipFree(d_mid_res_i);
	hipFree(d_final_res_i);

	

	
		
   return 0;
}

