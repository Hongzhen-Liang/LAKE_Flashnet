#include "hip/hip_runtime.h"
#include<stdio.h>
#include <stdbool.h> 
#include "weights.h"
#define LEN_INPUT 31
#define LEN_LAYER_0 256
#define LEN_LAYER_0_HALF 128
#define LEN_LAYER_1 2
#define FEAT_31


__global__ void prediction_mid_layer(long *weight_0_T_ent, long *bias_0_ent, long *input_vec_i, long *mid_res_i) { 
	int j, offset;

	int index = threadIdx.x;
    int stride = blockDim.x;

	for (j = index, offset=j*LEN_INPUT; j < LEN_LAYER_0; j+=stride, offset+=LEN_INPUT*stride) {
        mid_res_i[j] = 0;
        //loop unroll
		mid_res_i[j] =  mid_res_i[j] + input_vec_i[0] * weight_0_T_ent[offset+0]
		+ input_vec_i[1] * weight_0_T_ent[offset+1]
		+ input_vec_i[2] * weight_0_T_ent[offset+2]
		+ input_vec_i[3] * weight_0_T_ent[offset+3]
		+ input_vec_i[4] * weight_0_T_ent[offset+4]
		+ input_vec_i[5] * weight_0_T_ent[offset+5]
		+ input_vec_i[6] * weight_0_T_ent[offset+6]
		+ input_vec_i[7] * weight_0_T_ent[offset+7]
		+ input_vec_i[8] * weight_0_T_ent[offset+8]
		+ input_vec_i[9] * weight_0_T_ent[offset+9]
		+ input_vec_i[10] * weight_0_T_ent[offset+10]
		+ input_vec_i[11] * weight_0_T_ent[offset+11]
		+ input_vec_i[12] * weight_0_T_ent[offset+12]
		+ input_vec_i[13] * weight_0_T_ent[offset+13]
		+ input_vec_i[14] * weight_0_T_ent[offset+14]
		+ input_vec_i[15] * weight_0_T_ent[offset+15]
		+ input_vec_i[16] * weight_0_T_ent[offset+16]
		+ input_vec_i[17] * weight_0_T_ent[offset+17]
		+ input_vec_i[18] * weight_0_T_ent[offset+18]
		+ input_vec_i[19] * weight_0_T_ent[offset+19]
		+ input_vec_i[20] * weight_0_T_ent[offset+20]
		+ input_vec_i[21] * weight_0_T_ent[offset+21]
		+ input_vec_i[22] * weight_0_T_ent[offset+22]
		+ input_vec_i[23] * weight_0_T_ent[offset+23]
		+ input_vec_i[24] * weight_0_T_ent[offset+24]
		+ input_vec_i[25] * weight_0_T_ent[offset+25]
		+ input_vec_i[26] * weight_0_T_ent[offset+26]
		+ input_vec_i[27] * weight_0_T_ent[offset+27]
		+ input_vec_i[28] * weight_0_T_ent[offset+28]
		+ input_vec_i[29] * weight_0_T_ent[offset+29]
		+ input_vec_i[30] * weight_0_T_ent[offset+30];

        // apply bias
        mid_res_i[j] += bias_0_ent[j];
        // relu
        if (mid_res_i[j] < 0) {
            mid_res_i[j] = 0;
        }
    }
}

__global__ void prediction_final_layer(long *weight_1_T_ent, long *bias_1_ent, long *mid_res_i, long *final_res_i) {
    final_res_i[0] = 0;
	int k;
    for(k=0; k<LEN_LAYER_0; k ++) {
        final_res_i[0] =  final_res_i[0] + mid_res_i[k] * weight_1_T_ent[k];
	}
	// apply bias
	final_res_i[0] =  final_res_i[0] + bias_1_ent[0];

	final_res_i[1] = 0;
    for(k=0; k<LEN_LAYER_0; k ++) {
        final_res_i[1] =  final_res_i[1] + mid_res_i[k] * weight_1_T_ent[k+256];
	}
	// apply bias
	final_res_i[1] =  final_res_i[1] + bias_1_ent[1];
}

static bool prediction_model(long *d_input_vec_i, long *d_weight_0_T_ent, 
			long *d_weight_1_T_ent, long *d_bias_0_ent, long *d_bias_1_ent, long *d_mid_res_i, long *d_final_res_i) {

	long final_res_i[LEN_LAYER_1];

	prediction_mid_layer<<<1,256>>>(d_weight_0_T_ent, d_bias_0_ent, d_input_vec_i, d_mid_res_i);
	//hipDeviceSynchronize();
	//hipMemcpy(mid_res_i, d_mid_res_i, sizeof(long) * LEN_LAYER_0, hipMemcpyDeviceToHost);
	prediction_final_layer<<<1,1>>>(d_weight_1_T_ent, d_bias_1_ent, d_mid_res_i, d_final_res_i);

	hipMemcpy(final_res_i, d_final_res_i, sizeof(long) * 2, hipMemcpyDeviceToHost);
	// printf("\n%ld\n",final_res_i[1]);
	// printf("%ld\n",final_res_i[0]);
	return final_res_i[0]>=(final_res_i[1])? false: true;
}

int main() {
	long *weight_0_T_ent, * bias_0_ent, *weight_1_T_ent, * bias_1_ent; 
	long input_vec_i[31] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,0,0,0,0,9,0,0,0,9,0,0,0,9};

	weight_0_T_ent = &weight_i_0_T[0][0];
	weight_1_T_ent = &weight_i_1[0][0];
	bias_0_ent = bias_i_0;
	bias_1_ent = bias_i_1;

	long *d_weight_0_T_ent, *d_weight_1_T_ent, *d_bias_0_ent, *d_bias_1_ent, *d_input_vec_i, *d_mid_res_i, *d_final_res_i;

	hipMalloc((void**)&d_input_vec_i, sizeof(long) *LEN_INPUT);
	hipMalloc((void**)&d_weight_0_T_ent, sizeof(long) * 256*31);
	hipMalloc((void**)&d_weight_1_T_ent, sizeof(long) * 256*2);
	hipMalloc((void**)&d_bias_0_ent, sizeof(long) * 256);
	hipMalloc((void**)&d_bias_1_ent, sizeof(long) *2);

	hipMemcpy(d_weight_0_T_ent, weight_0_T_ent, sizeof(long) * 256*31, hipMemcpyHostToDevice);
	hipMemcpy(d_weight_1_T_ent, weight_1_T_ent, sizeof(long) * 256*2, hipMemcpyHostToDevice);
	hipMemcpy(d_bias_0_ent, bias_0_ent, sizeof(long) * 256, hipMemcpyHostToDevice);
	hipMemcpy(d_bias_1_ent, bias_1_ent, sizeof(long) * 2, hipMemcpyHostToDevice);

	hipMalloc((void**)&d_mid_res_i, sizeof(long) *LEN_LAYER_0);
	hipMalloc((void**)&d_final_res_i, sizeof(long) *LEN_LAYER_1);
	bool res;

	clock_t start = clock();
	for(int i = 0; i < 1000; i++) {
		hipMemcpy(d_input_vec_i, input_vec_i, sizeof(long) * LEN_INPUT, hipMemcpyHostToDevice);
		res = prediction_model(d_input_vec_i, d_weight_0_T_ent, 
			d_weight_1_T_ent, d_bias_0_ent, d_bias_1_ent, d_mid_res_i, d_final_res_i);
	}
	clock_t end = clock();
	float seconds = (float)(end - start) / CLOCKS_PER_SEC;
	printf("\n time taken : %f \n", seconds);

	hipFree(d_input_vec_i);
	hipFree(d_weight_0_T_ent);
	hipFree(d_weight_1_T_ent);
	hipFree(d_bias_0_ent);
	hipFree(d_bias_1_ent);
	hipFree(d_mid_res_i);
	hipFree(d_final_res_i);
	printf("\n %d", res);
		
   return 0;
}

