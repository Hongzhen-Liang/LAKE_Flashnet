#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include "weights.h"

__global__ void matrix_mult_constant(double *src, double constant, double *dest) {
    int blockId = blockIdx.x;
	int threadId = threadIdx.x;
	int dim = blockDim.x;
    dest[blockId*dim + threadId] = src[blockId*dim + threadId] * constant;
}

__global__ void matrix_add(double *src, double *add, double *dest) {
    int blockId = blockIdx.x;
	int threadId = threadIdx.x;
	int dim = blockDim.x;
    dest[blockId*dim + threadId] = src[blockId*dim + threadId] + add[blockId*dim + threadId];
}

__global__ void matrix_div_constant(double *src, double constant, double *dest) {
    int blockId = blockIdx.x;
	int threadId = threadIdx.x;
	int dim = blockDim.x;
    dest[blockId*dim + threadId] = src[blockId*dim + threadId] / constant;
}

__global__ void set_matrix_with_matrix(double *src, double *dest) { 
    int blockId = blockIdx.x;
	int threadId = threadIdx.x;
	int dim = blockDim.x;
    dest[blockId*dim + threadId] = src[blockId*dim + threadId];
}

__global__ void matrix_sub(double *src, double *sub, double *dest) { 
    int blockId = blockIdx.x;
	int threadId = threadIdx.x;
	int dim = blockDim.x;
    dest[blockId*dim + threadId] = src[blockId*dim + threadId] - sub[blockId*dim + threadId];
}

__global__ void matrix_elementwise_mult(double *m1, double *m2, double *dest) { 
    int blockId = blockIdx.x;
	int threadId = threadIdx.x;
	int dim = blockDim.x;
    dest[blockId*dim + threadId] = m1[blockId*dim + threadId] * m2[blockId*dim + threadId];
}

__global__ void matrix_elementwise_div(double *m1, double *m2, double *dest) { 
    int blockId = blockIdx.x;
	int threadId = threadIdx.x;
	int dim = blockDim.x;
    dest[blockId*dim + threadId] = m1[blockId*dim + threadId] / m2[blockId*dim + threadId];
}

__global__ void matrix_map(double *src, double (*func_f)(double), double *dest) { 
    int blockId = blockIdx.x;
	int threadId = threadIdx.x;
	int dim = blockDim.x;
    dest[blockId*dim + threadId] = func_f(src[blockId*dim + threadId]);
}

__global__ void matrix_transpose(double *m, double *ret) { 
    int blockId = blockIdx.x;
	int threadId = threadIdx.x;
	int dim = blockDim.x;
    ret[blockId*dim + threadId] = m[threadId * dim + blockId];
}

__global__ void matrix_repmat(double *m, int row_repeat, int col_repeat, int m_rows, int m_cols, double *ret) { 
    //int *ret = allocate(row_repeat*m_rows*col_repeat*m_cols);
    int blockId = blockIdx.x;
	int threadId = threadIdx.x;
	int dim = blockDim.x;
    if (col_repeat > 1) {
        for (int col_copy = 0; col_copy < col_repeat *m_cols; col_copy += m_cols) {
            ret[blockId*dim + (threadId +col_copy )] = m[blockId*dim + threadId];
        }
    }else {
        ret[blockId*dim + threadId] = m[blockId*dim + threadId];
    }
    if(row_repeat > 1) {
        for (int row_copy = m_rows; row_copy < m_rows*row_repeat; row_copy += m_rows) { 
            ret[(row_copy + blockId)*dim + threadId] = m[blockId*dim + threadId];
        }
    }
}

__global__ void matrix_mult(double *a,double *b, double *c, int m, int n, int k)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if( col < k && row < m) 
    {
        for(int i = 0; i < n; i++) 
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
}
