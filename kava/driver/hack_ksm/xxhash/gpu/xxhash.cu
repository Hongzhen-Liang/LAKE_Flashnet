#include "hip/hip_runtime.h"
#include <stdint.h>
#include <unistd.h>

// ============ Definitions ============
/*!
 * XXH_CPU_LITTLE_ENDIAN:
 * Defined to 1 if the target is little endian, or 0 if it is big endian.
 * It can be defined externally, for example on the compiler command line.
 *
 * If it is not defined, a runtime check (which is usually constant folded)
 * is used instead.
 */
#ifndef XXH_CPU_LITTLE_ENDIAN
/*
 * Try to detect endianness automatically, to avoid the nonstandard behavior
 * in `XXH_isLittleEndian()`
 */
#  if defined(_WIN32) /* Windows is always little endian */ \
     || defined(__LITTLE_ENDIAN__) \
     || (defined(__BYTE_ORDER__) && __BYTE_ORDER__ == __ORDER_LITTLE_ENDIAN__)
#    define XXH_CPU_LITTLE_ENDIAN 1
#  elif defined(__BIG_ENDIAN__) \
     || (defined(__BYTE_ORDER__) && __BYTE_ORDER__ == __ORDER_BIG_ENDIAN__)
#    define XXH_CPU_LITTLE_ENDIAN 0
#  else
static int XXH_isLittleEndian(void)
{
    /*
     * Nonstandard, but well-defined behavior in practice.
     * Don't use static: it is detrimental to performance.
     */
    const union { xxh_u32 u; xxh_u8 c[4]; } one = { 1 };
    return one.c[0];
}

#   define XXH_CPU_LITTLE_ENDIAN   XXH_isLittleEndian()
#  endif
#endif
#define XXH_rotl32(x,r) (((x) << (r)) | ((x) >> (32 - (r))))
// TODO: does little-endian stuff affect us?
#define XXH_get32bits(p) XXH_readLE32_align(p, align)
#define XXH_ASSERT(c)   assert(c)
typedef uint8_t  xxh_u8;
typedef uint32_t XXH32_hash_t;
typedef XXH32_hash_t xxh_u32;
typedef enum { XXH_aligned, XXH_unaligned } XXH_alignment;
#ifndef XXH_REROLL
#  if defined(__OPTIMIZE_SIZE__)
#    define XXH_REROLL 1
#  else
#    define XXH_REROLL 0
#  endif
#endif
#if defined(_MSC_VER)     /* Visual Studio */
#  define XXH_swap32 _byteswap_ulong
#elif XXH_GCC_VERSION >= 403
#  define XXH_swap32 __builtin_bswap32
#else
static xxh_u32 XXH_swap32 (xxh_u32 x)
{
    return  ((x << 24) & 0xff000000 ) |
            ((x <<  8) & 0x00ff0000 ) |
            ((x >>  8) & 0x0000ff00 ) |
            ((x >> 24) & 0x000000ff );
}
#endif

__device__ xxh_u32 XXH_readLE32_align(const void* ptr, XXH_alignment align)
{
  return XXH_CPU_LITTLE_ENDIAN ? *(const xxh_u32*)ptr : XXH_swap32(*(const xxh_u32*)ptr);
}


static const xxh_u32 PRIME32_1 = 0x9E3779B1U;   /* 0b10011110001101110111100110110001 */
static const xxh_u32 PRIME32_2 = 0x85EBCA77U;   /* 0b10000101111010111100101001110111 */
static const xxh_u32 PRIME32_3 = 0xC2B2AE3DU;   /* 0b11000010101100101010111000111101 */
static const xxh_u32 PRIME32_4 = 0x27D4EB2FU;   /* 0b00100111110101001110101100101111 */
static const xxh_u32 PRIME32_5 = 0x165667B1U;   /* 0b00010110010101100110011110110001 */

// ============ End definitions ============

/* mix all bits */
__device__ xxh_u32 XXH32_avalanche(xxh_u32 h32)
{
    h32 ^= h32 >> 15;
    h32 *= PRIME32_2;
    h32 ^= h32 >> 13;
    h32 *= PRIME32_3;
    h32 ^= h32 >> 16;
    return(h32);
}

__device__ xxh_u32 XXH32_finalize(xxh_u32 h32, const xxh_u8* ptr, size_t len, XXH_alignment align)
{
#define PROCESS1               \
    h32 += (*ptr++) * PRIME32_5; \
    h32 = XXH_rotl32(h32, 11) * PRIME32_1 ;

#define PROCESS4                         \
    h32 += XXH_get32bits(ptr) * PRIME32_3; \
    ptr+=4;                                \
    h32  = XXH_rotl32(h32, 17) * PRIME32_4 ;

    /* Compact rerolled version */
    if (XXH_REROLL) {
        len &= 15;
        while (len >= 4) {
            PROCESS4;
            len -= 4;
        }
        while (len > 0) {
            PROCESS1;
            --len;
        }
        return XXH32_avalanche(h32);
    } else {
         switch(len&15) /* or switch(bEnd - p) */ {
           case 12:      PROCESS4;
                         /* fallthrough */
           case 8:       PROCESS4;
                         /* fallthrough */
           case 4:       PROCESS4;
                         return XXH32_avalanche(h32);

           case 13:      PROCESS4;
                         /* fallthrough */
           case 9:       PROCESS4;
                         /* fallthrough */
           case 5:       PROCESS4;
                         PROCESS1;
                         return XXH32_avalanche(h32);

           case 14:      PROCESS4;
                         /* fallthrough */
           case 10:      PROCESS4;
                         /* fallthrough */
           case 6:       PROCESS4;
                         PROCESS1;
                         PROCESS1;
                         return XXH32_avalanche(h32);

           case 15:      PROCESS4;
                         /* fallthrough */
           case 11:      PROCESS4;
                         /* fallthrough */
           case 7:       PROCESS4;
                         /* fallthrough */
           case 3:       PROCESS1;
                         /* fallthrough */
           case 2:       PROCESS1;
                         /* fallthrough */
           case 1:       PROCESS1;
                         /* fallthrough */
           case 0:       return XXH32_avalanche(h32);
        }
        XXH_ASSERT(0);
        return h32;   /* reaching this point is deemed impossible */
    }
}

__device__ xxh_u32 XXH32_round(xxh_u32 acc, xxh_u32 input)
{
    acc += input * PRIME32_2;
    acc  = XXH_rotl32(acc, 13);
    acc *= PRIME32_1;
// TODO: look at this in more detail
#if 0 && defined(__GNUC__) && defined(__SSE4_1__) && !defined(XXH_ENABLE_AUTOVECTORIZE)
    /*
     * UGLY HACK:
     * This inline assembly hack forces acc into a normal register. This is the
     * only thing that prevents GCC and Clang from autovectorizing the XXH32
     * loop (pragmas and attributes don't work for some resason) without globally
     * disabling SSE4.1.
     *
     * The reason we want to avoid vectorization is because despite working on
     * 4 integers at a time, there are multiple factors slowing XXH32 down on
     * SSE4:
     * - There's a ridiculous amount of lag from pmulld (10 cycles of latency on
     *   newer chips!) making it slightly slower to multiply four integers at
     *   once compared to four integers independently. Even when pmulld was
     *   fastest, Sandy/Ivy Bridge, it is still not worth it to go into SSE
     *   just to multiply unless doing a long operation.
     *
     * - Four instructions are required to rotate,
     *      movqda tmp,  v // not required with VEX encoding
     *      pslld  tmp, 13 // tmp <<= 13
     *      psrld  v,   19 // x >>= 19
     *      por    v,  tmp // x |= tmp
     *   compared to one for scalar:
     *      roll   v, 13    // reliably fast across the board
     *      shldl  v, v, 13 // Sandy Bridge and later prefer this for some reason
     *
     * - Instruction level parallelism is actually more beneficial here because
     *   the SIMD actually serializes this operation: While v1 is rotating, v2
     *   can load data, while v3 can multiply. SSE forces them to operate
     *   together.
     *
     * How this hack works:
     * __asm__(""       // Declare an assembly block but don't declare any instructions
     *          :       // However, as an Input/Output Operand,
     *          "+r"    // constrain a read/write operand (+) as a general purpose register (r).
     *          (acc)   // and set acc as the operand
     * );
     *
     * Because of the 'r', the compiler has promised that seed will be in a
     * general purpose register and the '+' says that it will be 'read/write',
     * so it has to assume it has changed. It is like volatile without all the
     * loads and stores.
     *
     * Since the argument has to be in a normal register (not an SSE register),
     * each time XXH32_round is called, it is impossible to vectorize.
     */
    __asm__("" : "+r" (acc));
#endif
    return acc;
}

__device__ xxh_u32 XXH32_endian_align(const xxh_u8* input, size_t len, xxh_u32 seed, XXH_alignment align)
{
    const xxh_u8* bEnd = input + len;
    xxh_u32 h32;

#if defined(XXH_ACCEPT_NULL_INPUT_POINTER) && (XXH_ACCEPT_NULL_INPUT_POINTER>=1)
    if (input==NULL) {
        len=0;
        bEnd=input=(const xxh_u8*)(size_t)16;
    }
#endif

    if (len>=16) {
        const xxh_u8* const limit = bEnd - 15;
        xxh_u32 v1 = seed + PRIME32_1 + PRIME32_2;
        xxh_u32 v2 = seed + PRIME32_2;
        xxh_u32 v3 = seed + 0;
        xxh_u32 v4 = seed - PRIME32_1;

        do {
            v1 = XXH32_round(v1, XXH_get32bits(input)); input += 4;
            v2 = XXH32_round(v2, XXH_get32bits(input)); input += 4;
            v3 = XXH32_round(v3, XXH_get32bits(input)); input += 4;
            v4 = XXH32_round(v4, XXH_get32bits(input)); input += 4;
        } while (input < limit);

        h32 = XXH_rotl32(v1, 1)  + XXH_rotl32(v2, 7)
            + XXH_rotl32(v3, 12) + XXH_rotl32(v4, 18);
    } else {
        h32  = seed + PRIME32_5;
    }

    h32 += (xxh_u32)len;

    return XXH32_finalize(h32, input, len&15, align);
}

// TODO: for now, make len and seed deterministic
__global__ void XXH32(void *input, XXH32_hash_t *output)
{
  size_t len = 4096; 
  XXH32_hash_t seed = 17;
  int idx =
    blockIdx.x * blockDim.x + threadIdx.x +
    (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x + 
    (blockIdx.z * blockDim.z + threadIdx.z) * gridDim.x * blockDim.x * gridDim.y * blockDim.y;

  // Calculate the offset of the page to be hashed
  char *in_addr = ((char *) input) + idx * len;
  uint32_t *out_addr = ((uint32_t *) output) + idx;
#if 0
    /* Simple version, good for code maintenance, but unfortunately slow for small inputs */
    XXH32_state_t state;
    XXH32_reset(&state, seed);
    XXH32_update(&state, (const xxh_u8*)input, len);
    return XXH32_digest(&state);

#else
  *out_addr = XXH32_endian_align((const xxh_u8*) in_addr, len, seed, XXH_aligned);
#endif
}


// workspace must be 4 * (# of pages) * 4 bytes
// xxh_u32 v1 = 17 + PRIME32_1 + PRIME32_2;
// xxh_u32 v2 = 17 + PRIME32_2;
// xxh_u32 v3 = 17 + 0;
// xxh_u32 v4 = 17 - PRIME32_1;

__global__ void XXH32v2(void *input, XXH32_hash_t *output, uint32_t* workspace, uint32_t seed, uint32_t* seeds)
{
    size_t page_size = 4096; 
    //XXH32_hash_t seed = 17;
    int idx =
        blockIdx.x * blockDim.x + threadIdx.x +
        (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x + 
        (blockIdx.z * blockDim.z + threadIdx.z) * gridDim.x * blockDim.x * gridDim.y * blockDim.y;

    int page_offset = idx / 4;
    int word_offset = idx % 4;
    int ws_offset = (page_offset * 4) + word_offset;

    XXH_alignment align = XXH_aligned;

    // Calculate the offset of the page to be hashed
    char *page_addr = ((char *) input) + (page_offset * page_size);

    const xxh_u8* page_end = (const xxh_u8*) page_addr + page_size;
    const xxh_u8* const limit = page_end - 15;
    xxh_u32 v = seeds[word_offset]; 

    xxh_u8* thread_input = (xxh_u8*) (page_addr + (word_offset*4));
    do {
        v = XXH32_round(v, XXH_get32bits(thread_input)); 
        thread_input += 4;
    } while (thread_input < limit);
    workspace[ws_offset] = v;

    __syncthreads();

    if (word_offset == 0) {
        v = XXH_rotl32(workspace[ws_offset], 1)  + XXH_rotl32(workspace[ws_offset+1], 7)
            + XXH_rotl32(workspace[ws_offset+2], 12) + XXH_rotl32(workspace[ws_offset+3], 18);
        v += (xxh_u32)page_size;
        //finalize doesnt do anything on 4k pages
        //v = XXH32_finalize(h32, input, page_size&15, align);
        v = XXH32_avalanche(v);
        uint32_t *out_addr = ((uint32_t *) output) + page_offset;
        *out_addr = XXH32_endian_align((const xxh_u8*) page_addr, page_size, seed, XXH_aligned);
    }        
}

